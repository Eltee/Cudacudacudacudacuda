#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdlib.h"
#include "time.h"
#include "math.h"
#include "hip/hip_runtime.h"
#include ""

__global__ void CodeParallele(int n, int m, int np, double td, double h, float matDest) {










}


void codeSequentiel(int n, int m, int np, double td, double h) {

	printf("Lignes: %d \r\n", n);
	printf("Colonnes: %d \r\n", m);
	printf("Nombre de pas de temps: %d \r\n", np);
	printf("Temps discr�tis�: %f \r\n", td);
	printf("Taille d'un c�t� d'une subdivision: %f \r\n", h);

	int i, j, k;

	double*** plaque;
	plaque = new double**[n];
	for (int i = 0; i < n; i++) {
		plaque[i] = new double*[m];
		for (int j = 0; j < m; j++) {
			plaque[i][j] = new double[np];
		}
	}

	for (k = 0; k < np; k++) {
		for (i = 0; i < n; i++) {
			for (j = 0; j < m; j++) {
				plaque[i][j][k] = 0.0;
			}
		}
	}

	for (i = 0; i < n; i++) {
		for (j = 0; j < m; j++) {
			plaque[i][j][0] = i * (n - i - 1) * j * (m - j - 1);
		}
	}

	printf("Matrice initiale s�quentielle: \r\n");
	for (i = 0; i < n; i++) {
		for (j = 0; j < m; j++) {
			printf("%6.1f", plaque[i][j][0]);
		}
		printf("\r\n");
	}

	double timeStart, timeEnd, Texec;

	for (k = 1; k < np; k++) {
		for (i = n - 2; i >= 1; i--) {
			for (j = 1; j < m - 1; j++) {
				plaque[i][j][k] = ((1 - (4 * (td / (h*h)))) * plaque[i][j][k - 1]) + ((td / (h*h)) * (plaque[i - 1][j][k - 1] + plaque[i + 1][j][k - 1] + plaque[i][j - 1][k - 1] + plaque[i][j + 1][k - 1]));
				//usleep(5);
			}
		}
	}

	Texec = timeEnd - timeStart; //Temps d'ex�cution en secondes

	printf("Temps d'execution s�quentiel: %f \r\n", Texec);
	printf("Matrice finale s�quentielle: \r\n");
	for (i = 0; i < n; i++) {
		for (j = 0; j < m; j++) {
			printf("%6.1f", plaque[i][j][np - 1]);
		}
		printf("\r\n");
	}

	for (int i = 0; i < n; i++) {
		for (int j = 0; j < m; j++) {
			delete[] plaque[i][j];
		}
		delete[] plaque[i];
	}
	delete[] plaque;
}

int main(int argc, char *argv[])
{

	if (argc != 5)
	{
		printf("Pas assez d'arguments.\n");
		exit(1);
	}

	int n, m, np;
	double h, td;

	sscanf(argv[1], "%i", &n);
	sscanf(argv[2], "%i", &m);
	sscanf(argv[3], "%i", &np);
	td = atof(argv[4]);
	h = atof(argv[5]);

	//Sequentiel

	codeSequentiel(n, m, np, td, h);


	//Parallele
	double dim = sqrt(n*m);
	dim3 DimGrid(n, m, np);
	dim3 DimBlock(dim, dim, 1);
	float * mat;
	float * matDest;

	hipMalloc((void **)&mat, n*m *np * sizeof(float));
	hipMemcpy(matDest, mat, n*m * sizeof(float), hipMemcpyHostToDevice);


	CodeParallele<<<DimGrid, DimBlock>>>(n, m, np, td, h, matDest);


	hipFree(matDest);

}